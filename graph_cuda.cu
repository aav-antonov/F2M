#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <vector>
#include <stdlib.h>
#include <map>
#include <algorithm>    
#include <math.h>
#include <unordered_map>
#include <random>
#include <cmath>
#include <iomanip>      // std::setprecision
#include <cstring> 
#include <boost/algorithm/string/predicate.hpp>
//#################################################
#include <time.h>
#include <sys/time.h>
//#################################################
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h> 
#include <thrust/extrema.h>
//##########################################
//##########################################
#include "F2M.h"
#include "graph_cuda_define.hpp"
#include "graph_cuda_maxmin.hpp"
#include "graph_cuda_print.hpp"

#include "FXY.hpp"
#include "graph_tour.hpp"
#include "graph_cuda_init.hpp"

#include "graph_cuda_partition.hpp"
#include "graph_cuda_grad.hpp"
#include "graph_cuda_comb.hpp"



//#################################################
//#################################################
//#################################################

void graph::RUN(){


//#######################
dtimeRUN = dtime_usec(0);
//#######################
std::cout << "graph::RUN() " <<  std::endl;


error_B      = 1e-7; 
error_B_comb = 1e2;

graph_c * Pc = Ac;

Pc->error_B       = error_B;
Pc->error_B_comb  = error_B_comb ;

Pc->run_start_cuda();
Pc->get_combs_cuda(0 );
int IRR_B = host_combs_check();


int goshift = 0;
if(IRR_B > 0){goshift = 1;}

 double shiftV = 0.5;
 
 while(goshift == 1){ 
     
     Pc->copy_IRREGULAR_B_ToDevice();
     Pc->cuda_IRREGULAR_B_recompute(shiftV,0);
     
     Pc->free_combs_cuda();
     Pc->get_combs_cuda(1);  
     
     IRR_B = host_combs_check();
     
     if(IRR_B == 0){break;}
     
     Pc->copy_IRREGULAR_B_ToDevice();
     Pc->cuda_IRREGULAR_B_recompute(shiftV,1);
     
     Pc->free_combs_cuda();
     Pc->get_combs_cuda(1);  
     
     IRR_B = host_combs_check();
     
          
     if(IRR_B == 0){break;}
     
     shiftV = - shiftV;
     
     Pc->copy_IRREGULAR_B_ToDevice();
     Pc->cuda_IRREGULAR_B_recompute(shiftV,0);
     
     Pc->free_combs_cuda();
     Pc->get_combs_cuda(1);  
     
     IRR_B = host_combs_check();
     
     if(IRR_B == 0){break;}
     
     shiftV = - shiftV;
     
     Pc->copy_IRREGULAR_B_ToDevice();
     Pc->cuda_IRREGULAR_B_recompute(shiftV,1);
     
     Pc->free_combs_cuda();
     Pc->get_combs_cuda(1);  
     
     IRR_B = host_combs_check();
     
     if(IRR_B == 0){break;}
     
     shiftV = - shiftV;
     
     shiftV *= 2;
     
     if(fabs(shiftV) > 32){break;}  
 }



//#######################
dtimeRUN = dtime_usec(dtimeRUN);
std::cout << "dtimeRUN: " << dtimeRUN/(double)USECPSEC  << std::endl;
//#######################

 if(IRR_B == 0){
  for(int i=0; i < Vn; ++i ){V[i]->get_eA_X( this );}
  
  for(int i=0; i < Combs.size(); ++i ){
   for(auto jf : Combs[i]->XB){   
    X[jf.first] = Combs[i]->XB[jf.first];
    }
  }
     
  check_solution();
  print_file_solution(IRR_B);
  print_file_report(IRR_B);
  
  
 }else{
 
 print_file_report(IRR_B);
 
 }
 
}



//#################################################
//#################################################
//#################################################

void graph_c::run_start_cuda(){ 
 
  
  unsigned long long dtimeBG; 
  dtimeBG = dtime_usec(0);
 
  block_GLOB();

  
  std::cout << "block_GLOB ENDS " <<  std::endl;
   
  
  double M_grad =check_OPT();
  update_EF(1);
  
  int factor = 100;
  for(int go =1; go<= 10000; go++){ 
    
   block_EF(factor);
   double M_grad =check_OPT();
   update_EF(1);

   
   if(factor < 5000){factor += 300;}
   if(fabs(M_grad) < error_B){error_A = fabs(M_grad);break;}
  }
  
  
  
  dtimeBG = dtime_usec(dtimeBG);
  std::cout << "RUN dtime: " << dtimeBG/(double)USECPSEC  << std::endl;
  
  
  cudaCheckErrors("hipMemcpy fail run_start_cuda()");
  
  
}


  

void graph_c::run_fix_cuda(){ 
  
  unsigned long long dtimeBG; 
  
  dtimeBG = dtime_usec(0);
  
  double M_grad =check_OPT();
  update_EF(1);
  
  
  for(int go =1; go<= 100; go++){ 
    
   block_EF(500);
   double M_grad =check_OPT();
   update_EF(1);
   if(fabs(M_grad) < error_B){error_A = fabs(M_grad);break;}
  }
  
  
  
  dtimeBG = dtime_usec(dtimeBG);
  
} 


void graph_c::update_EF(int ind_to_partition ){ 
  
 
  if(ind_to_partition == -1){
   cuda_partition_v2<<<grid_size_Vn,block_size>>>(cuda_Edges_d_by_Nodes,  cuda_Nodes, cuda_NodesStat, cuda_EF_NodesStat, 1.00,Vn);
   hipDeviceSynchronize();
  }
  
    
  thrust::inclusive_scan(thrust::device,cuda_EF_NodesStat, cuda_EF_NodesStat + Vn, cuda_EF_Nodes);
  hipDeviceSynchronize();
  
  hipMemcpy(&size_EF, cuda_EF_Nodes + Vn-1, sizeof(int) , hipMemcpyDeviceToHost);
  //std::cout << "size_EF: " << size_EF  << std::endl;   
  
  get_cuda_EF_nodes_NORM<<<grid_size_Vn,block_size>>>( cuda_EF_NodesStat, cuda_EF_Nodes, Vn ); 
  hipDeviceSynchronize();
   
  hipFree(cuda_EF);
  hipMalloc((void**)&cuda_EF, sizeof(double**) * size_EF);
  
  get_cuda_EF<<<grid_size_Vn,block_size>>>(cuda_Edges_d_by_Nodes,  cuda_Nodes, cuda_EF_NodesStat, cuda_EF, cuda_EF_Nodes , Vn);
   
}

double graph_c::check_OPT(){ 
   
  cuda_recompute_Edges<<<grid_size_En,block_size>>>( cuda_Edges_d,  cuda_Edges_D,  cuda_Edges_v1,   cuda_Edges_v2, cuda_Nodes_F, En);
  hipDeviceSynchronize();
    
  cuda_partition_v2<<<grid_size_Vn,block_size>>>(cuda_Edges_d_by_Nodes,  cuda_Nodes, cuda_NodesStat, cuda_EF_NodesStat, 1.00,Vn);
  hipDeviceSynchronize();
  
  cuda_get_Nodes_grad_R<<<grid_size_Vn,block_size>>>(cuda_Edges_d_by_Nodes, cuda_Nodes, cuda_EF_NodesStat, cuda_Nodes_grad ,Vn) ;
  hipDeviceSynchronize();
  
  int iM_grad   = max_node_grad_S(cuda_Nodes_grad,  Vn);
  double M_grad = max_node_grad(cuda_Nodes_grad,  Vn);
  std::cout << "check_OPT() M_grad: " << M_grad << " " << iM_grad << std::endl;
  
  
return M_grad ;
} 
 
  

   


void graph_c::block_EF_iter_BLOCK(){ 
   
    cuda_get_Nodes_grad_R<<<grid_size_Vn,block_size>>>(cuda_EF, cuda_EF_Nodes, cuda_EF_NodesStat, cuda_Nodes_grad,Vn) ;
    hipDeviceSynchronize();
       
    cuda_update_Nodes_F_BLOCK<<<grid_size_Vn,block_size>>>(cuda_Nodes_grad,  cuda_Nodes_F ,  cuda_block_id,Vn);
    hipDeviceSynchronize();
    
    cuda_recompute_Edges<<<grid_size_En,block_size>>>(cuda_Edges_d,cuda_Edges_D,cuda_Edges_v1,cuda_Edges_v2,cuda_Nodes_F,En);
    hipDeviceSynchronize();
    
}      



void graph_c::block_EF_iter(){ 
    
    cuda_get_Nodes_grad_R<<<grid_size_Vn,block_size>>>(cuda_EF, cuda_EF_Nodes, cuda_EF_NodesStat, cuda_Nodes_grad,Vn);
    hipDeviceSynchronize();
    
    cuda_update_Nodes_F<<<grid_size_Vn,block_size>>>(cuda_Nodes_grad,  cuda_Nodes_F , Vn);
    hipDeviceSynchronize();
  
    cuda_recompute_Edges<<<grid_size_En,block_size>>>(cuda_Edges_d,cuda_Edges_D,cuda_Edges_v1,cuda_Edges_v2,cuda_Nodes_F,En);
    hipDeviceSynchronize();
    
}      



void graph_c::block_EF(int N){  
    
  cuda_get_Nodes_grad_R<<<grid_size_Vn,block_size>>>(cuda_EF, cuda_EF_Nodes, cuda_EF_NodesStat, cuda_Nodes_grad,Vn) ;
  hipDeviceSynchronize();
  
  double M_gradA = max_node_grad(cuda_Nodes_grad,  Vn);
  std::cout << "block_EF: M_gradA " << M_gradA  << "\n"; 
   
  int count_M = 0 ;//, count_stagnation = 0;
  
  for(int go = 0 ; go < N; go++){
    
    block_EF_iter();
   
    count_M++;
    if(count_M >= 50){
    
     
     int max_id = max_node_grad_S(cuda_Nodes_grad,  Vn);
     double M_gradB = max_node_grad(cuda_Nodes_grad,  Vn);
               
     if(fabs(M_gradA) < error_B){ std::cout << "block_EF: " << N << " go: " << go << " break\n";break;}
     
     count_M = 0;
    }
    
   }
   
  
  
}  

//##############################################################################
//##############################################################################
//##############################################################################



  

 

//##############################################################################
//##############################################################################
//##############################################################################

template <typename T>
__global__  void setvalue2V(T * v1, T * v2, T val, int n )
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n){
   if(v2[tid] > val){v1[tid]    =   val;}else{v1[tid]    =   v2[tid];}
  }
}

void graph_c::block_GLOB(){ 


  unsigned long long  dtimeBG; 
  
  dtimeBG = dtime_usec(0);
  
  int glob_K = 5;
  
  setvalue2V<<<grid_size_Vn,block_size>>>(cuda_EF_NodesStat, cuda_NodesStat, glob_K, Vn );
    
  
  for(int go = 0 ; go < 200; go++){
   
   block_GLOB_N(50);
   
   cuda_partition_v4<<<grid_size_Vn,block_size>>>(cuda_Edges_d_by_Nodes,  cuda_Nodes, cuda_NodesStat, cuda_EF_NodesStat, glob_K,Vn);
   hipDeviceSynchronize();
   
   //int max_pivot =  max_stat_v(cuda_EF_NodesStat, Vn);
   //int min_pivot =  min_stat_v(cuda_EF_NodesStat, Vn);
   //std::cout << "pivot: " << min_pivot << " " << max_pivot << std::endl;
   
   cuda_get_Nodes_grad_R<<<grid_size_Vn,block_size>>>(cuda_Edges_d_by_Nodes, cuda_Nodes, cuda_EF_NodesStat, cuda_Nodes_grad ,Vn) ;
   hipDeviceSynchronize();
   
   
   double M_grad = max_node_grad(cuda_Nodes_grad,  Vn);
   //std::cout << "block_GLOB_N: M_grad " << go << " " << M_grad  << std::endl;
   
   
    if(fabs(M_grad) < 0.01){
     std::cout << "block_GLOB_N: M_grad " << go << " " << M_grad  << std::endl;
     
     
    cuda_get_Nodes_grad_R<<<grid_size_Vn,block_size>>>(cuda_Edges_d_by_Nodes, cuda_Nodes, cuda_EF_NodesStat, cuda_Nodes_grad ,Vn) ;
    hipDeviceSynchronize();
  
    double M_grad = max_node_grad(cuda_Nodes_grad,  Vn);
    std::cout << "block_GLOB_N: M_grad " << go << " " << M_grad  << std::endl;
     
    break;
    }

   
   }
 
  dtimeBG = dtime_usec(dtimeBG);
  std::cout << "dtimeBG: " << dtimeBG/(double)USECPSEC  << std::endl;
     
}

void graph_c::block_GLOB_N(int N){  
  
    
  for(int go = 0 ; go < N; go++){
   
  cuda_get_Nodes_grad_R<<<grid_size_Vn,block_size>>>(cuda_Edges_d_by_Nodes, cuda_Nodes, cuda_EF_NodesStat, cuda_Nodes_grad ,Vn) ;
  hipDeviceSynchronize();
  
  cuda_update_Nodes_F<<<grid_size_Vn,block_size>>>(cuda_Nodes_grad,  cuda_Nodes_F ,Vn);
 
  cuda_recompute_Edges<<<grid_size_En,block_size>>>(cuda_Edges_d, cuda_Edges_D, cuda_Edges_v1, cuda_Edges_v2,cuda_Nodes_F, En );
  hipDeviceSynchronize();
  }
  
 
}









